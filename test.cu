
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main() {
    int n = 256;
    size_t size = n * sizeof(int);

    // allocate memory on CPU
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);
    int *h_c = (int*)malloc(size);

    // Init host arrays
    for (size_t i = 0; i < n; i++) {
        h_a[i] = i * 10;
        h_b[i] = i * 5;
    }

    // allocate memory on GPU
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // copy data from CPU to GPU
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // launch kernel on GPU with 256 threads
    add<<<1, n>>>(d_a, d_b, d_c);

    // copy data from GPU to CPU
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // verify the result
    size_t index = 100;
    if (h_c[index] == h_a[index] + h_b[index]) {
        std::cout << "Successful addition of vectors" << std::endl;
    } else {
        std::cout << "Failed" << std::endl;
    }

    // free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

